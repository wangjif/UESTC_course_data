#include "hip/hip_runtime.h"
#include <math.h>
#include <stdio.h>
#include <stdlib.h>
#include "timer.h"
#include "check.h"
#include <hip/hip_runtime.h>
//32 64 96 128 160 192 224 256
#define SOFTENING 1e-9f
#define BLOCK_SIZE 64
#define BLOCK_STRIDE 32
/*
 * Each body contains x, y, and z coordinate positions,
 * as well as velocities in the x, y, and z directions.
 */

typedef struct {
    float4 *pos, *vel;
} Body;

/*
 * Do not modify this function. A constraint of this exercise is
 * that it remain a host function.
 */

void randomizeBodies(float *data, int n) {
    for (int i = 0; i < n; i++) {
        data[i] = 2.0f * (rand() / (float) RAND_MAX) - 1.0f;
    }
}

/*
 * This function calculates the gravitational impact of all bodies in the system
 * on all others, but does not update their positions.
 */

__global__
void bodyForce(float4 *p, float4 *v, float dt, int n) {
    int id = threadIdx.x + blockDim.x * (int) (blockIdx.x / BLOCK_STRIDE);
    if (id < n) {
        int block_start = blockIdx.x % BLOCK_STRIDE;
        int total_block = n / BLOCK_SIZE;
        float Fx = 0.0f;
        float Fy = 0.0f;
        float Fz = 0.0f;
        float dx, dy, dz, distSqr, invDist, invDist3;
        float4 temp;
        __shared__ float3 spos[BLOCK_SIZE];
        float4 p_temp = p[id];
        for (int block_id = block_start; block_id < total_block; block_id += BLOCK_STRIDE) {
            //写入共享内存
            temp = p[block_id * BLOCK_SIZE + threadIdx.x];
            spos[threadIdx.x] = make_float3(temp.x, temp.y, temp.z);
            __syncthreads();

#pragma unroll 32
            for (int j = 0; j < BLOCK_SIZE; ++j) {
                dx = spos[j].x - p_temp.x;
                dy = spos[j].y - p_temp.y;
                dz = spos[j].z - p_temp.z;
                distSqr = dx * dx + dy * dy + dz * dz + SOFTENING;
                invDist = rsqrtf(distSqr);
                invDist3 = invDist * invDist * invDist;

                Fx += dx * invDist3;
                Fy += dy * invDist3;
                Fz += dz * invDist3;
            }
            __syncthreads();
        }
        atomicAdd(&v[id].x, dt * Fx);
        atomicAdd(&v[id].y, dt * Fy);
        atomicAdd(&v[id].z, dt * Fz);
    }
}

__global__ void change_pos(float4 *p, float4 *v, float dt, int n) {
    int i = threadIdx.x + blockDim.x * blockIdx.x;
    if (i < n) {
        p[i].x += v[i].x * dt;
        p[i].y += v[i].y * dt;
        p[i].z += v[i].z * dt;
    }
}

int main(const int argc, const char **argv) {

    /*
 * Do not change the value for `nBodies` here. If you would like to modify it,
 * pass values into the command line.
 */

    int nBodies = 2 << 11;
    int salt = 0;
    if (argc > 1) nBodies = 2 << atoi(argv[1]);

    /*
     * This salt is for assessment reasons. Tampering with it will result in automatic failure.
     */

    if (argc > 2) salt = atoi(argv[2]);

    const float dt = 0.01f; // time step
    const int nIters = 10;  // simulation iterations

    int bytes = 2 * nBodies * sizeof(float4);
    float *buf;

    hipHostMalloc(&buf, bytes);

    size_t Block_num = (nBodies + BLOCK_SIZE - 1) / BLOCK_SIZE;

    /*
  * As a constraint of this exercise, `randomizeBodies` must remain a host function.
  */

    float *temp_buf = (float *) malloc(nBodies * sizeof(float) * 6);
    randomizeBodies(temp_buf, 6 * nBodies); // Init pos / vel data
    //buf转换成我定义的格式
    int idx = 0;
    for (int i = 0; i < 4 * nBodies; i++) {
        if ((i + 1) % 4 == 0) {
            buf[i] = 0;
            idx += 3;
            continue;
        }
        buf[i] = temp_buf[idx++];
    }
    idx = 3;
    for (int i = 4 * nBodies; i < 8 * nBodies; i++) {
        if ((i + 1) % 4 == 0) {
            buf[i] = 0;
            idx += 3;
            continue;
        }
        buf[i] = temp_buf[idx++];
    }

    double totalTime = 0.0;

    float *device_buf;
    hipMalloc(&device_buf, bytes);
    Body d_p = {(float4 *) device_buf, ((float4 *) device_buf) + nBodies};
    hipMemcpy(device_buf, buf, bytes, hipMemcpyHostToDevice);

    /*
     * This simulation will run for 10 cycles of time, calculating gravitational
     * interaction amongst bodies, and adjusting their positions to reflect.
     */

    /*******************************************************************/
    // Do not modify these 2 lines of code.
    for (int iter = 0; iter < nIters; iter++) {
        StartTimer();
        /*******************************************************************/

        /*
         * You will likely wish to refactor the work being done in `bodyForce`,
         * as well as the work to integrate the positions.
         */

        bodyForce<<<Block_num * BLOCK_STRIDE, BLOCK_SIZE>>>(d_p.pos, d_p.vel, dt, nBodies); // compute interbody forces

        /*
         * This position integration cannot occur until this round of `bodyForce` has completed.
         * Also, the next round of `bodyForce` cannot begin until the integration is complete.
         */

        change_pos<<<Block_num, BLOCK_SIZE>>>(d_p.pos, d_p.vel, dt, nBodies);

        if (iter == nIters - 1)
//            hipDeviceSynchronize();
            hipMemcpy(buf, device_buf, bytes, hipMemcpyDeviceToHost);

        /*******************************************************************/
        // Do not modify the code in this section.
        const double tElapsed = GetTimer() / 1000.0;
        totalTime += tElapsed;
    }

    //把我的格式转换成老师的
    Body temp = {(float4 *) buf, ((float4 *) buf) + nBodies};
    for (int i = 0; i < nBodies; i++) {
        temp_buf[i * 6] = temp.pos[i].x;
        temp_buf[i * 6 + 1] = temp.pos[i].y;
        temp_buf[i * 6 + 2] = temp.pos[i].z;
        temp_buf[i * 6 + 3] = temp.vel[i].x;
        temp_buf[i * 6 + 4] = temp.vel[i].y;
        temp_buf[i * 6 + 5] = temp.vel[i].z;
    }
    buf = temp_buf;

    double avgTime = totalTime / (double) (nIters);
    float billionsOfOpsPerSecond = 1e-9 * nBodies * nBodies / avgTime;

#ifdef ASSESS
    checkPerformance(buf, billionsOfOpsPerSecond, salt);
#else
    checkAccuracy(buf, nBodies);
    printf("%d Bodies: average %0.3f Billion Interactions / second\n", nBodies, billionsOfOpsPerSecond);
//    printf("%0.3f\n", billionsOfOpsPerSecond);
    salt += 1;
#endif
    /*******************************************************************/

    /*
     * Feel free to modify code below.
     */

    hipFree(buf);
    hipFree(device_buf);
}