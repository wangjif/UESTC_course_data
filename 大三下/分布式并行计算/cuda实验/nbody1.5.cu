#include "hip/hip_runtime.h"
#include <math.h>
#include <stdio.h>
#include <stdlib.h>
#include "timer.h"
#include "check.h"
#include <hip/hip_runtime.h>
//32 64 96 128 160 192 224 256
#define SOFTENING 1e-9f
#define BLOCK_SIZE 64
#define BLOCK_STRIDE 32

/*
 * Each body contains x, y, and z coordinate positions,
 * as well as velocities in the x, y, and z directions.
 */

typedef struct {
    float x, y, z, vx, vy, vz;
} Body;

/*
 * Do not modify this function. A constraint of this exercise is
 * that it remain a host function.
 */

void randomizeBodies(float *data, int n) {
    for (int i = 0; i < n; i++) {
        data[i] = 2.0f * (rand() / (float) RAND_MAX) - 1.0f;
    }
}

/*
 * This function calculates the gravitational impact of all bodies in the system
 * on all others, but does not update their positions.
 */

__global__ void bodyForce(Body *p, float dt, int n) {
    //多出BLOCK_STRIDE倍的块进行计算
    int id = threadIdx.x + blockDim.x * (int) (blockIdx.x / BLOCK_STRIDE);
    if (id < n) {
        int block_start = blockIdx.x % BLOCK_STRIDE;
        int total_block = n / BLOCK_SIZE;
        float Fx = 0.0f;
        float Fy = 0.0f;
        float Fz = 0.0f;
        float dx, dy, dz, distSqr, invDist, invDist3;
        Body temp;
        __shared__ float3 spos[BLOCK_SIZE];
        Body p_temp = p[id];
        for (int block_id = block_start; block_id < total_block; block_id += BLOCK_STRIDE) {
            //写入共享内存
            temp = p[block_id * BLOCK_SIZE + threadIdx.x];
            spos[threadIdx.x] = make_float3(temp.x, temp.y, temp.z);
            __syncthreads();

#pragma unroll
            for (int j = 0; j < BLOCK_SIZE; ++j) {
                dx = spos[j].x - p_temp.x;
                dy = spos[j].y - p_temp.y;
                dz = spos[j].z - p_temp.z;
                distSqr = dx * dx + dy * dy + dz * dz + SOFTENING;
                invDist = rsqrtf(distSqr);
                invDist3 = invDist * invDist * invDist;
                Fx += dx * invDist3;
                Fy += dy * invDist3;
                Fz += dz * invDist3;
            }
            __syncthreads();
        }
        atomicAdd(&p[id].vx, dt * Fx);
        atomicAdd(&p[id].vy, dt * Fy);
        atomicAdd(&p[id].vz, dt * Fz);
    }
}

__global__ void change_pos(Body *p, float dt, int n) {
    int i = threadIdx.x + blockDim.x * blockIdx.x;
    if (i < n) {
        p[i].x += p[i].vx * dt;
        p[i].y += p[i].vy * dt;
        p[i].z += p[i].vz * dt;
    }
}

int main(const int argc, const char **argv) {

    /*
     * Do not change the value for `nBodies` here. If you would like to modify it,
     * pass values into the command line.
     */

    int nBodies = 2 << 11;
    int salt = 0;
    if (argc > 1) nBodies = 2 << atoi(argv[1]);

    /*
     * This salt is for assessment reasons. Tampering with it will result in automatic failure.
     */

    if (argc > 2) salt = atoi(argv[2]);

    const float dt = 0.01f; // time step
    const int nIters = 10;  // simulation iterations

    int bytes = nBodies * sizeof(Body);
    float *buf;

    hipHostMalloc(&buf, bytes);

    size_t Block_num = (nBodies + BLOCK_SIZE - 1) / BLOCK_SIZE;

    /*
     * As a constraint of this exercise, `randomizeBodies` must remain a host function.
     */

    randomizeBodies(buf, 6 * nBodies); // Init pos / vel data

    double totalTime = 0.0;

    float *device_buf;
    hipMalloc(&device_buf, bytes);
    Body *device_p = (Body *) device_buf;
    hipMemcpy(device_buf, buf, bytes, hipMemcpyHostToDevice);

    /*
     * This simulation will run for 10 cycles of time, calculating gravitational
     * interaction amongst bodies, and adjusting their positions to reflect.
     */

    /*******************************************************************/
    // Do not modify these 2 lines of code.
    for (int iter = 0; iter < nIters; iter++) {
        StartTimer();
        /*******************************************************************/

        /*
         * You will likely wish to refactor the work being done in `bodyForce`,
         * as well as the work to integrate the positions.
         */

        bodyForce<<<Block_num * BLOCK_STRIDE, BLOCK_SIZE>>>(device_p, dt, nBodies); // compute interbody forces

        /*
         * This position integration cannot occur until this round of `bodyForce` has completed.
         * Also, the next round of `bodyForce` cannot begin until the integration is complete.
         */

        change_pos<<<Block_num, BLOCK_SIZE>>>(device_p, dt, nBodies);

        if (iter == nIters - 1)
//            hipDeviceSynchronize();
            hipMemcpy(buf, device_buf, bytes, hipMemcpyDeviceToHost);

        /*******************************************************************/
        // Do not modify the code in this section.
        const double tElapsed = GetTimer() / 1000.0;
        totalTime += tElapsed;
    }

    double avgTime = totalTime / (double) (nIters);
    float billionsOfOpsPerSecond = 1e-9 * nBodies * nBodies / avgTime;

#ifdef ASSESS
    checkPerformance(buf, billionsOfOpsPerSecond, salt);
#else
    checkAccuracy(buf, nBodies);
    printf("%d Bodies: average %0.3f Billion Interactions / second\n", nBodies, billionsOfOpsPerSecond);
//    printf("%0.3f\n", billionsOfOpsPerSecond);
    salt += 1;
#endif
    /*******************************************************************/

    /*
     * Feel free to modify code below.
     */

    hipFree(buf);
    hipFree(device_buf);
}
